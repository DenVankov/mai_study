
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>


#define HANDLE_ERROR(err)                             \
    do { if (err != hipSuccess) { printf("ERROR: %s\n", hipGetErrorString(err)); exit(0);} } while (0)


__global__ void diffVector(double *v1, double *v2, double *res, int size)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = gridDim.x * blockDim.x;
    while (idx < size) {
        res[idx] = v1[idx] - v2[idx];
        idx += offset;
    }
}


int main()
{
    std::ios_base::sync_with_stdio(false);

    int size = 0;
    std::cin >> size;

    double *vec1 = new double[size];
    double *vec2 = new double[size];
    double *res = new double[size];

    for (int i = 0; i < size; ++i) {
        std::cin >> vec1[i];
    }
    for (int i = 0; i < size; ++i) {
        std::cin >> vec2[i];
    }

    double *dev1, *dev2, *devRes;

    HANDLE_ERROR(hipMalloc((void **) &dev1, sizeof(double) * size));
    HANDLE_ERROR(hipMalloc((void **) &dev2, sizeof(double) * size));
    HANDLE_ERROR(hipMalloc((void **) &devRes, sizeof(double) * size));

    HANDLE_ERROR(hipMemcpy(dev1, vec1, sizeof(double) * size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev2, vec2, sizeof(double) * size, hipMemcpyHostToDevice));

    diffVector<<<256, 256>>>(dev1, dev2, devRes, size);
    HANDLE_ERROR(hipGetLastError());

    HANDLE_ERROR(hipMemcpy(res, devRes, sizeof(double) * size, hipMemcpyDeviceToHost));

    std::cout.precision(10);
    std::cout.setf(std::ios::scientific);
    for (int i = 0; i < size; ++i) {
        std::cout << res[i] << ' ';
    }
    std::cout << '\n';

    HANDLE_ERROR(hipFree(dev1));
    HANDLE_ERROR(hipFree(dev2));
    HANDLE_ERROR(hipFree(devRes));

    delete[] vec1;
    delete[] vec2;
    delete[] res;
}
