
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>


#define HANDLE_ERROR(err)                             \
    do { if (err != hipSuccess) { printf("ERROR: %s\n", hipGetErrorString(err)); exit(0);} } while (0)

__constant__ double AVG[32][3];
__constant__ double COV[32][3][3];
__constant__ double COV_INV[32][3][3];
__constant__ double DETS[32];


__device__ double func(uchar4 p, int i)
{
    double res = 0.0, p_avg[3], tmp[3];
    for (int j = 0; j < 3; ++j) {
        p_avg[j] = 0.0;
        tmp[j] = 0.0;
    }
    p_avg[0] = p.x - AVG[i][0];
    p_avg[1] = p.y - AVG[i][1];
    p_avg[2] = p.z - AVG[i][2];

    for (int j = 0; j < 3; ++j) {
        for (int k = 0; k < 3; ++k) {
            tmp[j] += -p_avg[k] * COV_INV[i][k][j];
        }
        res += tmp[j] * p_avg[j];
    }

    res -= std::log(std::abs(DETS[i]));
    return res;
}

__device__ int getClass(uchar4 pixel, int nc)
{
    double res[32];
    for (int i = 0; i < nc; ++i) {
        res[i] = func(pixel, i);
    }
    double maxEl = res[0];
    int idx = 0;
    for (int i = 0; i < nc; ++i) {
        if (res[i] > maxEl) {
            maxEl = res[i];
            idx = i;
        }
    }
    return idx;
}


__global__ void kernel(uchar4 *dst, int w, int h, int nc)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    for (int x = idx; x < w; x += offsetx) {
        for (int y = idy; y < h; y += offsety) {
            dst[x + y * w].w = getClass(dst[x + y * w], nc);
        }
    }
}


int main()
{
    std::string input, output;
    int w, h, nc, np;
    uchar4 *data;
    std::cin >> input >> output >> nc;
    std::vector<std::vector<int2>> classes(nc);
    for (int i = 0; i < nc; ++i) {
        std::cin >> np;
        classes[i].resize(np);
        for (int j = 0; j < np; ++j) {
            std::cin >> classes[i][j].x >> classes[i][j].y;
        }
    }

    std::ifstream fsIn(input, std::ios::in | std::ios::binary);
    if (fsIn.is_open()) {
        fsIn.read((char *)&w, sizeof(w));
        fsIn.read((char *)&h, sizeof(h));
        data = new uchar4[w * h];
        fsIn.read((char *)data, w * h * sizeof(data[0]));
        fsIn.close();
    } else {
        return 1;
    }

    // дальше начинаются интересные вещи
    double avg[32][3];
    for (int i = 0; i < 32; ++i) {
        for (int j = 0; j < 3; ++j) {
            avg[i][j] = 0.0;
        }
    }

    for (int i = 0; i < nc; ++i) {
        int np = classes[i].size();
        for (int j = 0; j < np; ++j) {
            int x = classes[i][j].x;
            int y = classes[i][j].y;
            uchar4 curPixel = data[x + y * w];
            avg[i][0] += curPixel.x;
            avg[i][1] += curPixel.y;
            avg[i][2] += curPixel.z;
        }
        for (int k = 0; k < 3; ++k) {
            avg[i][k] /= np;
        }
    }

    double cov[32][3][3];
    for (int i = 0; i < 32; ++i) {
        for (int j = 0; j < 3; ++j) {
            for (int k = 0; k < 3; ++k) {
                cov[i][j][k] = 0.0;
            }
        }
    }

    for (int i = 0; i < nc; ++i) {
        np =  classes[i].size();
        for (int j = 0; j < np; ++j) {
            double tmp[3];
            int x = classes[i][j].x;
            int y = classes[i][j].y;
            uchar4 curPixel = data[x + y * w];
            tmp[0] = curPixel.x - avg[i][0];
            tmp[1] = curPixel.y - avg[i][1];
            tmp[2] = curPixel.z - avg[i][2];

            for (int k = 0; k < 3; ++k) {
                for (int l = 0; l < 3; ++l) {
                    cov[i][k][l] += tmp[k] * tmp[l];
                }
            }
        }
        for (int k = 0; k < 3; ++k) {
            for (int l = 0; l < 3; ++l) {
                cov[i][k][l] /= np - 1;
            }
        }
    }

    double cov_inv[32][3][3];
    for (int i = 0; i < 32; ++i) {
        for (int j = 0; j < 3; ++j) {
            for (int k = 0; k < 3; ++k) {
                cov_inv[i][j][k] = 0.0;
            }
        }
    }

    double dets[32];
    for (int i = 0; i < nc; ++i) {
        double det = 0;
        for (int j = 0; j < 3; ++j) {
            det += cov[i][0][j] * (cov[i][1][(j + 1) % 3] * cov[i][2][(j + 2) % 3] - cov[i][1][(j + 2) % 3] * cov[i][2][(j + 1) % 3]);
        }
        dets[i] = det;
    }

    // извините
    for (int i = 0; i < nc; ++i) {
        cov_inv[i][0][0] = (cov[i][1][1] * cov[i][2][2] - cov[i][2][1] * cov[i][1][2]) / dets[i];
        cov_inv[i][0][1] = (cov[i][0][2] * cov[i][2][1] - cov[i][0][1] * cov[i][2][2]) / dets[i];
        cov_inv[i][0][2] = (cov[i][0][1] * cov[i][1][2] - cov[i][0][2] * cov[i][1][1]) / dets[i];
        cov_inv[i][1][0] = (cov[i][1][2] * cov[i][2][0] - cov[i][1][0] * cov[i][2][2]) / dets[i];
        cov_inv[i][1][1] = (cov[i][0][0] * cov[i][2][2] - cov[i][0][2] * cov[i][2][0]) / dets[i];
        cov_inv[i][1][2] = (cov[i][1][0] * cov[i][0][2] - cov[i][0][0] * cov[i][1][2]) / dets[i];
        cov_inv[i][2][0] = (cov[i][1][0] * cov[i][2][1] - cov[i][2][0] * cov[i][1][1]) / dets[i];
        cov_inv[i][2][1] = (cov[i][2][0] * cov[i][0][1] - cov[i][0][0] * cov[i][2][1]) / dets[i];
        cov_inv[i][2][2] = (cov[i][0][0] * cov[i][1][1] - cov[i][1][0] * cov[i][0][1]) / dets[i];
    }

    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(AVG), avg, sizeof(double) * 32 * 3));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(COV), cov, sizeof(double) * 32 * 3 * 3));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(COV_INV), cov_inv, sizeof(double) * 32 * 3 * 3));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(DETS), dets, sizeof(double) * 32));

    uchar4 *dev_data;
    HANDLE_ERROR(hipMalloc(&dev_data, sizeof(uchar4) * h * w));
    HANDLE_ERROR(hipMemcpy(dev_data, data, sizeof(uchar4) * h * w, hipMemcpyHostToDevice));
    kernel<<<dim3(16, 16), dim3(16, 16)>>>(dev_data, w, h, nc);
    HANDLE_ERROR(hipMemcpy(data, dev_data, sizeof(uchar4) * h * w, hipMemcpyDeviceToHost));

    std::ofstream fsOut(output, std::ios::out | std::ios::binary);
    if (fsOut.is_open()) {
        fsOut.write((char *)&w, sizeof(w));
        fsOut.write((char *)&h, sizeof(h));
        fsOut.write((char *)data, w * h * sizeof(data[0]));
        fsOut.close();
    } else {
        return 1;
    }

    HANDLE_ERROR(hipFree(dev_data));

    delete[] data;
    return 0;
}