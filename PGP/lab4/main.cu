#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>

#include <thrust/extrema.h>
#include <thrust/device_vector.h>


#define HANDLE_ERROR(err)                             \
    do { if (err != hipSuccess) { printf("ERROR: %s\n", hipGetErrorString(err)); exit(0);} } while (0)


struct comparator
{
    __host__ __device__ bool operator()(double a, double b)
    {
        return std::fabs(a) < std::fabs(b);
    }
};


__global__ void swapRows(double *data, int n, int i, int i_max)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;

    for (int j = idx; j < n; j += offset) {
        double tmp = data[j * n + i];
        data[j * n + i] = data[j * n + i_max];
        data[j * n + i_max] = tmp;
    }
}


__global__ void divide(double *data, int n, int i)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;

    for (int j = idx + i + 1; j < n; j += offset) {
        data[i * n + j] /= data[i * n + i];
    }
}


__global__ void kernel(double *data, int n, int i)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    for (int j = idx + i + 1; j < n; j += offsetx) {
        for (int k = idy + i + 1; k < n; k += offsety) {
            data[k * n + j] -= data[i * n + j] * data[k * n + i];
        }
    }
}


int main()
{
    std::ios_base::sync_with_stdio(false);
    std::cin.tie(nullptr);
    int n;
    std::cin >> n;
    double *mat = new double[n * n];
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cin >> mat[i + j * n];
        }
    }

    int *p = new int[n];
    for (int i = 0; i < n; ++i) {
        p[i] = i;
    }

    double *data;
    HANDLE_ERROR(hipMalloc((void **) &data, sizeof(double) * n * n));
    HANDLE_ERROR(hipMemcpy(data, mat, sizeof(double) * n * n, hipMemcpyHostToDevice));

    comparator comp;
    thrust::device_ptr<double> i_ptr, i_max_ptr;
    for (int i = 0; i < n - 1; ++i) {
        int i_max = i;
        i_ptr = thrust::device_pointer_cast(data + i * n);
        i_max_ptr = thrust::max_element(i_ptr + i, i_ptr + n, comp);
        i_max = i_max_ptr - i_ptr;

        if (i_max != i) {
            p[i] = i_max;
            swapRows<<<256, 256>>>(data, n, i, i_max);
        }

        divide<<<256, 256>>>(data, n, i);
        kernel<<<dim3(32, 32), dim3(32, 32)>>>(data, n, i);
    }
    hipMemcpy(mat, data, sizeof(double) * n * n, hipMemcpyDeviceToHost);

    std::cout.precision(10);
    std::cout.setf(std::ios::scientific);
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cout << mat[i + j * n] << " ";
        }
        std::cout << "\n";
    }

    for (int i = 0; i < n; ++i) {
        std::cout << p[i] << " ";
    }
    std::cout << "\n";

    HANDLE_ERROR(hipFree(data));
    delete[] mat;
    delete[] p;
    return 0;
}